#include "hip/hip_runtime.h"
#include <convexa.h>
#include <cxkernels.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#define BLOCK_LEN 1024
namespace CXKernels
{
    template <typename T = double>
    __global__ void overlap_save_full_convolve(T *A, T *B, T *C, unsigned int aN, unsigned int bN, unsigned int cN)
    {
        extern __shared__ __align__(sizeof(T)) unsigned char my_smem[];
        T *s_sig = reinterpret_cast<T *>(my_smem);
        unsigned int w_offset = bN / 2;
        unsigned int sig_size = bN + BLOCK_LEN ;
        // printf("%d Size!", sig_size);

        unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        for (int n = index; n < ((cN / BLOCK_LEN) + 1) * BLOCK_LEN; n += blockDim.x * gridDim.x) // scuttles down the signal
        {
            for (int sig_i = threadIdx.x; sig_i < sig_size ; sig_i+= blockDim.x) {
                int sig_index =  n - bN + sig_i - threadIdx.x;
                if (sig_index >= 0 && sig_index < aN) { s_sig[sig_i] = A[sig_index];}
                else s_sig[sig_i] = 0;
            }
            __syncthreads();

            if (n < cN)
            {
                T sum = 0.0;
                // Convert the pixel
                for (int i = 0; i < bN; i += 1) // scuttles down the kernel
                {
                    sum += B[ i] * s_sig[bN   +threadIdx.x - i];//* A[n - i];
                }
                C[n] = sum;
            }
        }
    }
}
template <typename T = double>
float CXTiming::device_convolve_overlap_save(const std::vector<T> &signal, const std::vector<T> &kernel, std::vector<T> &output)
{
    T *device_a = nullptr;
    T *device_b = nullptr;
    T *device_c = nullptr;
    size_t byte_size_sig = signal.size() * sizeof(T);
    size_t byte_size_kernel = kernel.size() * sizeof(T);
    size_t ol = (signal.size() + kernel.size() - 1);
    size_t byte_size_output = ol * sizeof(T);
    output.resize(ol);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&device_a, byte_size_sig));
    checkCudaErrors(hipMemcpy(device_a, signal.data(), byte_size_sig, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_b, byte_size_kernel));
    checkCudaErrors(hipMemcpy(device_b, kernel.data(), byte_size_kernel, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_c, byte_size_output));
    // We don't actually need to do any copying, as a matter of fact, that could result in stinky.
    // checkCudaErrors(hipMemcpy(device_c, C, byte_size, hipMemcpyHostToDevice));

    dim3 blockSize(BLOCK_LEN);
    if (ol < BLOCK_LEN)
        blockSize.x = ol;
    int blocks = signal.size() / blockSize.x + 1;
    dim3 gridSize(blocks);
    size_t shmem = (BLOCK_LEN + kernel.size() )* sizeof(T);
    hipEventRecord(start);
    // Memory Loaded, Perform Computations...
    CXKernels::overlap_save_full_convolve<T><<<gridSize, blockSize, shmem>>>(device_a, device_b, device_c, signal.size(), kernel.size(), ol);

    hipEventRecord(stop);

    // Finish Computations before this block
    checkCudaErrors(hipMemcpy(output.data(), device_c, byte_size_output, hipMemcpyDeviceToHost));
    hipGetLastError();

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_b));
    checkCudaErrors(hipFree(device_c));

    return milliseconds;
}

template float CXTiming::device_convolve_overlap_save<double>(const std::vector<double> &, const std::vector<double> &, std::vector<double> &);
template float CXTiming::device_convolve_overlap_save<float>(const std::vector<float> &, const std::vector<float> &, std::vector<float> &);
// template float CXTiming::device_convolve_overlap_add<uint16_t>(const std::vector<uint16_t>&, const std::vector<uint16_t>&, std::vector<uint16_t>&);
// template float CXTiming::device_convolve_overlap_add<int16_t>(const std::vector<int16_t>&, const std::vector<int16_t>&, std::vector<int16_t>&);