#include "hip/hip_runtime.h"
#include <convexa.h>
#include <cxkernels.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
typedef enum CX_Conv_Type {OVERLAP_ADD, FFT_BASED} CXConv_t;

#define DEBUG false

template <typename T>
struct conv_ptrs
{
    hipStream_t stream;
    CXConv_t conv_type;
    std::vector<T*> device_ptrs;
    std::vector<hipfftComplex*> device_fft_ptrs;
    std::vector<hipfftHandle> plans;
};
template <>
struct conv_ptrs<double>
{
    hipStream_t stream;
    CXConv_t conv_type;
    std::vector<double*> device_ptrs;
    std::vector<hipfftDoubleComplex*> device_fft_ptrs;
    std::vector<hipfftHandle> plans;
};

template< typename T >
conv_ptrs<T> choose_convolution(const std::vector<T> &signal, const std::vector<T> &kernel)
{
    size_t signal_length = signal.size();
    size_t kernel_length = kernel.size();
    size_t classic_bigO = kernel_length * (kernel_length + signal_length - 1);
    size_t fftconv_bigO = 3 * signal_length * log2(signal_length) + signal_length;

    conv_ptrs<T> group;
    hipStream_t stream{0};
    checkCudaErrors(hipStreamCreate(&stream));
    group.stream = stream;

    T* d_signal, *d_kernel, *d_result;

    if (classic_bigO < fftconv_bigO)
    {
        group.conv_type = OVERLAP_ADD;
        
        checkCudaErrors(hipMalloc(&d_signal, signal_length * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_kernel, kernel_length * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_result, (signal_length + kernel_length - 1) * sizeof(T)));

        checkCudaErrors(hipMemcpyAsync(d_signal, signal.data(), signal_length * sizeof(T),
                        hipMemcpyHostToDevice, group.stream));
        checkCudaErrors(hipMemcpyAsync(d_kernel, kernel.data(), kernel_length * sizeof(T),
                        hipMemcpyHostToDevice, group.stream));

        group.device_ptrs.push_back(d_signal);
        group.device_ptrs.push_back(d_kernel);
        group.device_ptrs.push_back(d_result);
    }
    else
    {
        group.conv_type = FFT_BASED;
        
        hipfftComplex* d_signal_fft, *d_kernel_fft, *d_result_fft;
        uint32_t fft_size = 1;
        while (fft_size < (signal_length + kernel_length - 1)) fft_size <<= 1;
        
        checkCudaErrors(hipMalloc(&d_signal, fft_size * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_kernel, fft_size * sizeof(T)));
        checkCudaErrors(hipMalloc(&d_result, fft_size * sizeof(T)));

        checkCudaErrors(hipMemset(d_signal, 0, fft_size * sizeof(T)));
        checkCudaErrors(hipMemset(d_kernel, 0, fft_size * sizeof(T)));

        checkCudaErrors(hipMemcpyAsync(d_signal, signal.data(), signal_length * sizeof(T),
                        hipMemcpyHostToDevice, group.stream));
        checkCudaErrors(hipMemcpyAsync(d_kernel, kernel.data(), kernel_length * sizeof(T),
                        hipMemcpyHostToDevice, group.stream));


        group.device_ptrs.push_back(d_signal);
        group.device_ptrs.push_back(d_kernel);
        group.device_ptrs.push_back(d_result);

        checkCudaErrors(hipMalloc(&d_signal_fft, fft_size * sizeof(hipfftComplex)));
        checkCudaErrors(hipMalloc(&d_kernel_fft, fft_size * sizeof(hipfftComplex)));
        checkCudaErrors(hipMalloc(&d_result_fft, fft_size * sizeof(hipfftComplex)));
        group.device_fft_ptrs.push_back(d_signal_fft);
        group.device_fft_ptrs.push_back(d_kernel_fft);
        group.device_fft_ptrs.push_back(d_result_fft);

        hipfftHandle planForward, planInverse;
        hipfftPlan1d(&planForward, fft_size, HIPFFT_R2C, 1);
        hipfftPlan1d(&planInverse, fft_size, HIPFFT_C2R, 1);
        group.plans.push_back(planForward);
        group.plans.push_back(planInverse);
    }

    return group;
}

template <typename T>
void launch_convolution(conv_ptrs<T> group, size_t signal_length, size_t kernel_length)
{
    
    checkCudaErrors(hipStreamSynchronize(group.stream));

    uint32_t result_length = (signal_length + kernel_length - 1);

    if (group.conv_type == OVERLAP_ADD)
    {
        dim3 blockSize(1024);
        int blocks = signal_length / blockSize.x + 1;
        dim3 gridSize(blocks);
        size_t shmem = (1024 + kernel_length) * sizeof(T);
        CXKernels::overlap_save_full_convolve<T> <<< gridSize, blockSize, shmem, group.stream >>> (
            group.device_ptrs[0], group.device_ptrs[1], group.device_ptrs[2], 
            signal_length, kernel_length, result_length
        );

    } else if (group.conv_type == FFT_BASED) {

        dim3 blockSize(1024);
        uint32_t fft_size = 1;
        while (fft_size < result_length) fft_size <<= 1;
        if (fft_size < 1024)
            blockSize.x = fft_size;
        int blocks = signal_length / blockSize.x + 1;
        dim3 gridSize(blocks);
        size_t shmem = (1024 + kernel_length) * sizeof(T);

        hipfftExecR2C(group.plans[0], group.device_ptrs[0], group.device_fft_ptrs[0]);
        hipfftExecR2C(group.plans[0], group.device_ptrs[1], group.device_fft_ptrs[1]);

        CXKernels::vec_multiply_complex_f <<< gridSize, blockSize, 0, group.stream >>> (
            group.device_fft_ptrs[0], group.device_fft_ptrs[1], group.device_fft_ptrs[2], fft_size
        );

        hipfftExecC2R(group.plans[1], group.device_fft_ptrs[2], group.device_ptrs[2]);
    }
}

template <typename T>
std::vector<std::vector<T>> ConvExa::batch_convolve(const std::vector<std::vector<T>> &signals, const std::vector<std::vector<T>> &kernels)
{
    std::vector<std::vector<T>> results;
    std::vector<conv_ptrs<T>> workspace;

    // Load the memory
    for (uint32_t idx = 0; idx < signals.size(); idx++)
    {
        conv_ptrs<T> group = choose_convolution(signals[idx], kernels[idx]);
        workspace.push_back(group);
    }
    if (DEBUG) {
        for (conv_ptrs<T> group: workspace) {
            printf("Call Type: %d \n", group.conv_type);
            // printf("", )
        }
    }

    for (uint32_t idx = 0; idx < signals.size(); idx++)
    {
        launch_convolution(workspace[idx], signals[idx].size(), kernels[idx].size());
    }

    for (uint32_t idx = 0; idx < signals.size(); idx++)
    {
        uint32_t sig_length = signals[idx].size();
        uint32_t ker_length = kernels[idx].size();
        uint32_t result_length = (sig_length + ker_length - 1);
        uint32_t result_size = result_length * sizeof(T);
        
        std::vector<T> result; 
        result.resize(result_length);
        results.push_back(result);
        // Wait for convolution to complete
        checkCudaErrors(hipStreamSynchronize(workspace[idx].stream));
        checkCudaErrors(hipMemcpyAsync(results[idx].data(), workspace[idx].device_ptrs[2], result_size,
                        hipMemcpyDeviceToHost, workspace[idx].stream));

        if (workspace[idx].conv_type == FFT_BASED) {
            uint32_t fft_size = 1;
            while (fft_size < result_length) fft_size <<= 1;
            T scale = 1.0 / static_cast<T>(fft_size);
            for (int i = 0; i < results[idx].size(); i ++) {
                results[idx][i]  *= scale;
            }
        }

        // Memory copy is complete
        checkCudaErrors(hipStreamSynchronize(workspace[idx].stream));
    }
    for (auto &group : workspace)
    {
        for (auto &ptr : group.device_ptrs)
        {
            checkCudaErrors(hipFree(ptr));
        }
        for (auto &ptr : group.device_fft_ptrs)
        {
            checkCudaErrors(hipFree(ptr));
        }
        for (auto &plan : group.plans)
        {
            hipfftDestroy(plan);
        }
    }

    checkCudaErrors(hipDeviceSynchronize());
    return results;
}
template std::vector<std::vector<float>> ConvExa::batch_convolve(const std::vector<std::vector<float>> &signals, const std::vector<std::vector<float>> &kernels);