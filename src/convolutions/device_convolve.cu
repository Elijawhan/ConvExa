#include "hip/hip_runtime.h"
#include <convexa.h>
#include <cxkernels.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
namespace CXKernels
{
    template <typename T = double>
    __global__ void basic_full_convolve(T *A, T *B, T *C, unsigned int aN, unsigned int bN, unsigned int cN)
    {
        unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
        for (int n = index; n < cN; n += blockDim.x * gridDim.x) // scuttles down the signal
        {
            T sum = 0.0;
            // Convert the pixel
            for (int i = 0; i < bN; i += 1) // scuttles down the kernel
            {
                if (n >= i && (n - i) < aN)
                    sum += B[i] * A[n - i];
            }
            C[n] = sum;
        }
    }
}
template <typename T= double>
float CXTiming::device_convolve(const std::vector<T> &signal, const std::vector<T> &kernel, std::vector<T> &output)
{
    T *device_a = nullptr;
    T *device_b = nullptr;
    T *device_c = nullptr;
    size_t byte_size_sig = signal.size() * sizeof(T);
    size_t byte_size_kernel = kernel.size() * sizeof(T);
    size_t ol = (signal.size() + kernel.size() - 1);
    size_t byte_size_output = ol * sizeof(T);
    output.resize(ol);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&device_a, byte_size_sig));
    checkCudaErrors(hipMemcpy(device_a, signal.data(), byte_size_sig, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_b, byte_size_kernel));
    checkCudaErrors(hipMemcpy(device_b, kernel.data(), byte_size_kernel, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_c, byte_size_output));
    // We don't actually need to do any copying, as a matter of fact, that could result in stinky.
    // checkCudaErrors(hipMemcpy(device_c, C, byte_size, hipMemcpyHostToDevice));


    dim3 blockSize(1024);
    if (ol < 1024) blockSize.x = ol;
    int blocks = signal.size() / blockSize.x + 1;
    dim3 gridSize(blocks);
    hipEventRecord(start);
    // Memory Loaded, Perform Computations...
    CXKernels::basic_full_convolve<T><<<gridSize, blockSize>>>(device_a, device_b, device_c, signal.size(), kernel.size(), ol );

    hipEventRecord(stop);

    // Finish Computations before this block
    checkCudaErrors(hipMemcpy(output.data(), device_c, byte_size_output, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_b));
    checkCudaErrors(hipFree(device_c));

    return milliseconds;
}

template float CXTiming::device_convolve<double>(const std::vector<double>&, const std::vector<double>&, std::vector<double>&);
template float CXTiming::device_convolve<uint16_t>(const std::vector<uint16_t>&, const std::vector<uint16_t>&, std::vector<uint16_t>&);
template float CXTiming::device_convolve<int16_t>(const std::vector<int16_t>&, const std::vector<int16_t>&, std::vector<int16_t>&);