#include "hip/hip_runtime.h"
#include <convexa.h>
#include <cxkernels.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

namespace CXKernels
{

// Device-side constants used in DFT
// Try moving to constant memory?
__device__ constexpr cuda::std::complex<double> j_d(0.0, 1.0);
__device__ constexpr cuda::std::complex<float> j_f(0.0, 1.0);

template <typename T>
__global__ void device_dft(const T* signal, const size_t length, cuda::std::complex<T>* result)
{
    // Grid-stride over the signal. Each thread receives its own DFT component.
    for (uint32_t index = blockDim.x*blockIdx.x + threadIdx.x;
         index < length;
         index += blockDim.x * gridDim.x)
    {
        // Thread calculates local sum
        cuda::std::complex<T> sum(0.0, 0.0);
        for (uint32_t i = 0; i < length; i++)
        {
            // Use CUDA Std Lib to create complex component
            cuda::std::complex<T> exponential = cuda::std::exp(
                -(j_d * 2.0 * HIP_PI * 
                 static_cast<T>(index) * static_cast<T>(i) / static_cast<T>(length))
            );
            // Add to sum
            sum += static_cast<cuda::std::complex<T>>(signal[i]) * exponential;
        }
        // Store sum back into result
        result[index] = sum;
    }
}
// Explicit float template specialization 
// (originally wanted to call cuda::std::expf, but expf does not accept complex inputs)
template <>
__global__ void device_dft(const float* signal, const size_t length, cuda::std::complex<float>* result)
{
    for (uint32_t index = blockDim.x*blockIdx.x + threadIdx.x;
         index < length;
         index += blockDim.x * gridDim.x)
    {
        cuda::std::complex<float> sum(0.0, 0.0);
        for (uint32_t i = 0; i < length; i++)
        {
            cuda::std::complex<float> exponential = cuda::std::exp(
                -(j_f * 2.0f * HIP_PI_F * 
                 static_cast<float>(index) * static_cast<float>(i) / static_cast<float>(length))
            );
            sum += static_cast<cuda::std::complex<float>>(signal[i]) * exponential;
        }
        result[index] = sum;
    }
}
}

template <typename T>
float CXTiming::device_dft(const std::vector<T> &signal, std::vector<std::complex<T>> &result)
{
    T* device_a = nullptr;
    cuda::std::complex<T>* device_c = nullptr;
    uint32_t length = signal.size();

    size_t byte_size_sig = length * sizeof(T);
    size_t byte_size_output = length * sizeof(std::complex<T>);
    result.resize(length);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&device_a, byte_size_sig));
    checkCudaErrors(hipMemcpy(device_a, signal.data(), byte_size_sig, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_c, byte_size_output));

    dim3 num_threads = 1024;
    dim3 num_blocks = (length + num_threads.x - 1) / num_threads.x;
    hipEventRecord(start);
    // Memory Loaded, Perform Computations...
    CXKernels::device_dft<<<num_blocks, num_threads>>>(
        device_a, length, device_c
    );
    hipEventRecord(stop);

    // Finish Computations before this block
    checkCudaErrors(hipMemcpy(result.data(), device_c, byte_size_output, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_c));

    T scale = 1.0 / static_cast<T>(length);
    for (int i = 0; i < result.size(); ++i) {
        result[i] *= length;
    }

    return milliseconds;
}
template float CXTiming::device_dft<double>(const std::vector<double> &signal, std::vector<std::complex<double>> &result);
template float CXTiming::device_dft<float>(const std::vector<float> &signal, std::vector<std::complex<float>> &result);