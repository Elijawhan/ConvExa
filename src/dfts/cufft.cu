#include <convexa.h>
#include <cxkernels.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <helper.h>

template <typename T>
float CXTiming::cufft(const std::vector<T> &signal, std::vector<std::complex<T>> &result)
{
    std::vector<std::complex<T>> signal_complex = HELP::vec_cast<T, std::complex<T>>(signal);

    hipfftComplex* device_a = nullptr;
    hipfftComplex* device_c = nullptr;
    uint32_t length = signal.size();

    size_t byte_size = length * sizeof(hipfftComplex);
    result.resize(length);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&device_a, byte_size));
    // C++ standard, I apologize sincerely for this.
    checkCudaErrors(hipMemcpy(device_a, reinterpret_cast<hipfftComplex*>(signal_complex.data()), byte_size, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_c, byte_size));

    hipfftHandle plan;
    hipfftPlan1d(&plan, length, HIPFFT_C2C, 1);
    hipEventRecord(start);
    hipfftExecC2C(plan, device_a, device_c, HIPFFT_FORWARD);
    hipEventRecord(stop);

    // Finish Computations before this block
    checkCudaErrors(hipMemcpy(result.data(), reinterpret_cast<std::complex<T>*>(device_c), byte_size, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_c));
    hipfftDestroy(plan);

    T scale = 1.0 / static_cast<T>(length);
    for (int i = 0; i < result.size(); ++i) {
        result[i] *= length;
    }

    return milliseconds;
}
template float CXTiming::cufft<float>(const std::vector<float> &signal, std::vector<std::complex<float>> &result);

template <>
float CXTiming::cufft<double>(const std::vector<double> &signal, std::vector<std::complex<double>> &result)
{
    std::vector<std::complex<double>> signal_complex = HELP::vec_cast<double, std::complex<double>>(signal);

    hipfftDoubleComplex* device_a = nullptr;
    hipfftDoubleComplex* device_c = nullptr;
    uint32_t length = signal.size();

    size_t byte_size = length * sizeof(hipfftDoubleComplex);
    result.resize(length);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    checkCudaErrors(hipMalloc(&device_a, byte_size));
    // C++ standard, I apologize sincerely for this.
    checkCudaErrors(hipMemcpy(device_a, reinterpret_cast<hipfftDoubleComplex*>(signal_complex.data()), byte_size, hipMemcpyHostToDevice));

    checkCudaErrors(hipMalloc(&device_c, byte_size));

    hipfftHandle plan;
    hipfftPlan1d(&plan, length, HIPFFT_Z2Z, 1);
    hipEventRecord(start);
    hipfftExecZ2Z(plan, device_a, device_c, HIPFFT_FORWARD);
    hipEventRecord(stop);

    // Finish Computations before this block
    checkCudaErrors(hipMemcpy(result.data(), reinterpret_cast<std::complex<double>*>(device_c), byte_size, hipMemcpyDeviceToHost));

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    checkCudaErrors(hipFree(device_a));
    checkCudaErrors(hipFree(device_c));
    hipfftDestroy(plan);

    double scale = 1.0 / static_cast<double>(length);
    for (int i = 0; i < result.size(); ++i) {
        result[i] *= length;
    }
    
    return milliseconds;
}
